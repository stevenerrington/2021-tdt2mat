/**
 * block loading rho calculation. should be much faster
 * system('nvcc -ptx citydist_rho4.cu')
 * iA is multiple of CHUNK (16)
*/

#include <hip/hip_runtime.h>
// #include "cublas_v2.h"
#include <math.h>
#define ABS(my_val) ((my_val) < 0) ? (-1*(my_val)) : (my_val)
#define MIN(A,B) ((A)<(B)) ? (A) : (B)
#define MAX(A,B) ((A)>(B)) ? (A) : (B)
#define NTHREADS 128
#define NC (1+6*2) // number of Channels
#define CHUNK 16 //previously defined as CHUNK
#define SINGLE_INF (3.402E+38)

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 * Step through one B at a time
 */
__global__ void citydist_sorted_delta(float const *A, unsigned int const *I, float *D, unsigned int *N, int const nA, int const nneigh, int const nC){
    // int iA = blockIdx.x * CHUNK;    
    int iA = (blockIdx.x + blockIdx.y * gridDim.x) * CHUNK;
    int iA1;
    int tx = threadIdx.x;
    float vr_minDist1[CHUNK];
    unsigned int vi_minIdx1[CHUNK];
    __shared__ unsigned int svi_I_A1[CHUNK];
    __shared__ float smr_A1[NC][CHUNK];
    __shared__ float smr_delta1[NTHREADS][CHUNK];
    __shared__ unsigned int smi_nneigh1[NTHREADS][CHUNK]; 
    
    // cache A
    int iC = tx;    
    if (iC < nC){ //use tx as iC
        for (iA1 = 0; iA1 < CHUNK; ++iA1){
            if (iA + iA1 < nA){
                smr_A1[iC][iA1] = A[iC + (iA+iA1)*nC];
            }else{
                smr_A1[iC][iA1] = SINGLE_INF;
            }   
        }
    }
    
    iA1 = tx; // batch index
    if (iA1 < CHUNK){
        if (iA + iA1 < nA){
            svi_I_A1[iA1] = I[iA + iA1];  
        }else{
            svi_I_A1[iA1] = nA + 1;  // out of range
        }
    }
    for (iA1 = 0; iA1 < CHUNK; ++iA1){
        vr_minDist1[iA1] = SINGLE_INF;
        vi_minIdx1[iA1] = iA + iA1;
    }    
    __syncthreads();

    // fill in the shared memory A
    int iB_min = MAX(iA - nneigh, 0);
    int iB_max = MIN(iA + nneigh + CHUNK - 1, nA-1);
    if (nneigh==0){
        iB_min = 0; 
        iB_max = nA-1;
    }      
    int iB = iB_min + tx;  
    while (iB <= iB_max){
        float vr_dist1[CHUNK];
        for (iA1 = 0; iA1 < CHUNK; ++iA1) vr_dist1[iA1] = 0.0f;
        for (iC = 0; iC < nC; ++iC){
            float Btemp = A[iC + iB*nC];
            for (iA1 = 0; iA1 < CHUNK; ++iA1){
                float temp = Btemp - smr_A1[iC][iA1];
                vr_dist1[iA1] += ABS(temp);
            }            
        }          
        unsigned int IiB = I[iB];
        for (iA1 = 0; iA1 < CHUNK; ++iA1){            
            if (vr_dist1[iA1] < vr_minDist1[iA1]){
                if (IiB < svi_I_A1[iA1]){                
            //if (vr_dist1[iA1] < vr_minDist1[iA1] && vr_dist1[iA1]>0){
                //if (IiB < svi_I_A1[iA1] && iB != iA+iA1){
                    int dab = ABS(iA + iA1 - iB);
                    if (dab <= nneigh || nneigh==0){
                        vr_minDist1[iA1] = vr_dist1[iA1];
                        vi_minIdx1[iA1] = iB;
                    }
                }
            }
        }
        iB += blockDim.x;
    } // while
    
    // collect result from each thread
    for (iA1 = 0; iA1 < CHUNK; ++iA1){        
        smr_delta1[tx][iA1] = vr_minDist1[iA1];
        smi_nneigh1[tx][iA1] = vi_minIdx1[iA1];
    }
    __syncthreads();    
    
    // final count    
    iA1 = tx;
    if (iA1 < CHUNK && iA + iA1 < nA){
        float minDist1 = SINGLE_INF;
        unsigned int minIdx1 = iA + iA1;
        for (int tx1=0; tx1<blockDim.x; ++tx1){
            if (smr_delta1[tx1][iA1] < minDist1){
                minDist1 = smr_delta1[tx1][iA1];
                minIdx1 = smi_nneigh1[tx1][iA1];
            }
        }
        D[iA + iA1] = sqrtf(minDist1);
        N[iA + iA1] = minIdx1;        
    }

} // func