/**
 * jrc_cuda_rho.cu
 * block loading rho calculation. should be much faster
 * system('nvcc -ptx -m 64 -arch sm_35 jrc_cuda_rho.cu')
 * i1 is multiple of chunk (16)
 * J. James Jun, Vidrio Technologies, LLC., 2017 Jun 11
 * 7/13/17: fDc_spk option added, which uses spike-specific distance cut-off (dc)
*/

#include <hip/hip_runtime.h>
#include <math.h>
#define ABS(my_val) ((my_val) < 0) ? (-1*(my_val)) : (my_val)
#define MIN(A,B) ((A)<(B)) ? (A) : (B)
#define MAX(A,B) ((A)>(B)) ? (A) : (B)
#define NTHREADS 128
#define MAXDIM 45
#define CHUNKSIZE 16
#define SINGLE_INF (3.402E+38) // equivalent to NAN. consider -1 value

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 * Step through one B at a time
 */
__global__ void jrc_cuda_rho(float *rho, const float *site_features, const int *spike_order, const int *site_constants, const float dist_cut2) {
    int i1 = (blockIdx.x + blockIdx.y * gridDim.x) * CHUNKSIZE; // base index of i1
    int thread_x = threadIdx.x; // nThreadsGPU for i12 index
    int i1_thread_x = i1 + thread_x;
    int n_spikes_primary = site_constants[0];
    int n_spikes_all = site_constants[1];
    int n_features = site_constants[2];
    int dn_max = site_constants[3];
    int fDc_spk = site_constants[4];

    __shared__ int spike_order_chunk[CHUNKSIZE];
    __shared__ float features_primary[MAXDIM][CHUNKSIZE];
    __shared__ int rho_chunk[NTHREADS][CHUNKSIZE]; // count then divide later
    __shared__ int mnComp1_[NTHREADS][CHUNKSIZE]; // count number of elements compared
    __shared__ float vrDc1_[CHUNKSIZE];  // use if fDc_spk=1

    // cache shared memory
    if (thread_x < n_features) {
        for (int i_c = 0; i_c < CHUNKSIZE; i_c++) {
            int i1_c = i_c + i1;
            if (i1_c < n_spikes_primary) {
                features_primary[thread_x][i_c] = site_features[thread_x + i1_c * n_features];
            } else {
                features_primary[thread_x][i_c] = 0.0f;
            }
        }
    }

    if (thread_x < CHUNKSIZE && i1_thread_x < n_spikes_primary) {
        spike_order_chunk[thread_x] = spike_order[i1_thread_x];
    }

    // initialize rho
    for (int i_c = 0; i_c < CHUNKSIZE; i_c++) {
        rho_chunk[thread_x][i_c] = 0;
        mnComp1_[thread_x][i_c] = 0;
    }

    // calculate spike-specific distance cut-off vrDc1_ only if fDc_spk==1
    if (thread_x < CHUNKSIZE && fDc_spk == 1) {
        vrDc1_[thread_x] = 0.0f; //init
        for (int i_feature = 0; i_feature < n_features; i_feature++) {
            float temp = features_primary[i_feature][thread_x];
            vrDc1_[thread_x] += (temp * temp);
        }
        vrDc1_[thread_x] *= dist_cut2;
    }

    __syncthreads();

    // Inspect distance relationship between i1 and i12_tx
    for (int i12_tx = thread_x; i12_tx < n_spikes_all; i12_tx += blockDim.x) {
        int iiSpk12_ord_tx = spike_order[i12_tx];

        // compute distance
        float feature_dists2_chunk[CHUNKSIZE]; // square of pairwise feature distances for chunk
        for (int i_c = 0; i_c < CHUNKSIZE; i_c++) {
            feature_dists2_chunk[i_c] = 0.0f;
        }

        for (int i_feature = 0; i_feature < n_features; i_feature++) {
            float fet12_tx = site_features[i_feature + i12_tx * n_features];
            for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
                float temp = fet12_tx - features_primary[i_feature][i_c]; // z_i = x_i - y_i
                feature_dists2_chunk[i_c] += temp * temp;                 // dist += z_i^2
            }
        }

        // Compare the index and distance
        for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
            int time_dist = ABS(spike_order_chunk[i_c] - iiSpk12_ord_tx);
            if (time_dist <= dn_max) {
                ++mnComp1_[thread_x][i_c];
                if (fDc_spk == 0) {
                    if (feature_dists2_chunk[i_c] <= dist_cut2) {
                        ++rho_chunk[thread_x][i_c];
                    }
                } else {
                    if (feature_dists2_chunk[i_c] < vrDc1_[i_c]) {
                        ++rho_chunk[thread_x][i_c];
                    }
                }
            }
        }
    } // for

    // final count
    __syncthreads();

    if (thread_x < CHUNKSIZE) {  // use thread_x as i_c
        int nRho1 = 0;
        int nComp1 = 0;
        for (int tx1 = 0; tx1 < blockDim.x; tx1++) {
            nRho1 += rho_chunk[tx1][thread_x];
            nComp1 += mnComp1_[tx1][thread_x];
        }

        if (i1_thread_x < n_spikes_primary) {
            rho[i1_thread_x] = (float)(((double) (nRho1)) / ((double) nComp1));
        }
    }
}