/**
 * jrc_cuda_delta.cu
 * block loading delta calculation. should be much faster
 * system('nvcc -ptx -m 64 -arch sm_35 jrc_cuda_rho.cu')
 * iA is multiple of CHUNKSIZE (16)
 * J. James Jun, Vidrio Technologies, LLC., 2017 Jun 11
*/

#include <hip/hip_runtime.h>
#include <math.h>
#define ABS(my_val) ((my_val) < 0) ? (-1*(my_val)) : (my_val)
#define MIN(A,B) ((A)<(B)) ? (A) : (B)
#define MAX(A,B) ((A)>(B)) ? (A) : (B)
#define NTHREADS 128
#define MAXDIM 45 // number of Channels
#define CHUNKSIZE 16 // previously defined as CHUNK
#define SINGLE_INF (3.402E+38)

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 * Step through one B at a time
 * 7/13/17: fDc_spk option added, which uses spike-specific distance cut-off (dc)
 */
__global__ void jrc_cuda_delta(float *delta, unsigned int *nneigh, const float *site_features, const int *spike_order, const int *rho_order, const int *site_constants, const float dist_cut2) {
    int i1 = (blockIdx.x + blockIdx.y * gridDim.x) * CHUNKSIZE; // base index of i1
    int thread_x = threadIdx.x;
    int i1_thread_x = i1 + thread_x;
    int n_spikes_primary = site_constants[0];
    int n_spikes_all = site_constants[1];
    int n_features = site_constants[2];
    int time_dist_cut = site_constants[3];
    int fDc_spk = site_constants[4];

    __shared__ int spike_order_chunk[CHUNKSIZE];
    __shared__ int rho_order_chunk[CHUNKSIZE];
    __shared__ float features_primary[MAXDIM][CHUNKSIZE];
    __shared__ float mrDelta1_[NTHREADS][CHUNKSIZE];
    __shared__ unsigned int miNneigh1_[NTHREADS][CHUNKSIZE];
    __shared__ float vrDc1_[CHUNKSIZE]; // use if fDc_spk == 1

    // cache shared memory, 1/2
    if (thread_x < n_features) { // use thread_x as iC
        for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
            int i1_c = i_c + i1;
            if (i1_c < n_spikes_primary) {
                features_primary[thread_x][i_c] = site_features[thread_x + i1_c * n_features];
            } else {
                features_primary[thread_x][i_c] = 0.0f;
            }
        }
    }

    // cache shared memory, 2/2
    if (thread_x < CHUNKSIZE && i1_thread_x < n_spikes_primary) {
        spike_order_chunk[thread_x] = spike_order[i1_thread_x];
        rho_order_chunk[thread_x] = rho_order[i1_thread_x];
    }

    float mindist_chunk[CHUNKSIZE];
    unsigned int nneigh_chunk[CHUNKSIZE];
    for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
        mindist_chunk[i_c] = SINGLE_INF;
        nneigh_chunk[i_c] = i1 + i_c; // self
    }

    // calculate spike-specific distance cut-off vrDc1_ only if fDc_spk == 1
    if (thread_x < CHUNKSIZE && fDc_spk == 1) {
        vrDc1_[thread_x] = 0.0f; // init
        for (int iC = 0; iC < n_features; ++iC) {
            float temp_ = features_primary[iC][thread_x];
            vrDc1_[thread_x] += (temp_ * temp_);
        }
        vrDc1_[thread_x] *= dist_cut2;
    }

    __syncthreads();

    // fill in the shared memory A
    for (int i12_tx = thread_x; i12_tx < n_spikes_all; i12_tx += blockDim.x) {
        // compute time difference
        char nearby_in_time[CHUNKSIZE];
        int i_spike_order = spike_order[i12_tx];
        int i_rho_order = rho_order[i12_tx];

        for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
            char rho_is_larger = (i_rho_order < rho_order_chunk[i_c]); // is rho larger?
            int time_dist = ABS(spike_order_chunk[i_c] - i_spike_order); // is the spike nearby in time?
            nearby_in_time[i_c] = (time_dist <= time_dist_cut) && rho_is_larger;
        }

        // compute distance
        float feature_dists2_chunk[CHUNKSIZE]; // square of pairwise feature distances for chunk
        for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
            feature_dists2_chunk[i_c] = 0.0f;
        }

        for (int iC = 0; iC < n_features; ++iC) {
            float fet12_tx = site_features[iC + i12_tx * n_features];
            for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
                float temp = fet12_tx - features_primary[iC][i_c]; // z_i = x_i - y_i
                feature_dists2_chunk[i_c] += temp * temp;          // dist += z_i^2
            }
        }

        // Compare the index and distance
        for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
            if ((nearby_in_time[i_c] == 1) && (feature_dists2_chunk[i_c] < mindist_chunk[i_c])) {
                mindist_chunk[i_c] = feature_dists2_chunk[i_c];
                nneigh_chunk[i_c] = i12_tx;
            }
        }
    } // for

    // collect result from each thread
    for (int i_c = 0; i_c < CHUNKSIZE; ++i_c) {
        mrDelta1_[thread_x][i_c] = mindist_chunk[i_c];
        miNneigh1_[thread_x][i_c] = nneigh_chunk[i_c];
    }
    __syncthreads();

    // final count
    if (thread_x < CHUNKSIZE) {
        float minDist1 = SINGLE_INF;
        unsigned int minIdx1 = i1_thread_x;
        for (int tx1 = 0; tx1 < blockDim.x; ++tx1) {
            if (mrDelta1_[tx1][thread_x] < minDist1) {
                minDist1 = mrDelta1_[tx1][thread_x];
                minIdx1 = miNneigh1_[tx1][thread_x];
            }
        }

        if (i1_thread_x < n_spikes_primary) {
            if (fDc_spk == 0) {
                delta[i1_thread_x] = sqrtf(ABS(minDist1) / dist_cut2);
            } else {
                delta[i1_thread_x] = sqrtf(ABS(minDist1) / vrDc1_[thread_x]);
            }
            nneigh[i1_thread_x] = minIdx1 + 1; // Matlab index output
        }
    }
} // func