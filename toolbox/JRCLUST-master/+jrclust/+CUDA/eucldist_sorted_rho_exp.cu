/**
 * block loading rho calculation. should be much faster
 * system('nvcc -ptx citydist_rho4.cu')
 * iA is multiple of chunk (16)
*/

#include <hip/hip_runtime.h>
// #include "cublas_v2.h"
#include <math.h>
#define ABS(my_val) ((my_val) < 0) ? (-1*(my_val)) : (my_val)
#define MIN(A,B) ((A)<(B)) ? (A) : (B)
#define MAX(A,B) ((A)>(B)) ? (A) : (B)
#define NTHREADS 128
#define NC (1+6*2)
// #define NC (9)
#define CHUNK 16
#define SINGLE_INF (3.402E+38)

/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 * Step through one B at a time
 */
__global__ void eucldist_sorted_rho_exp(float const *A, float *D, int const nA, int const nneigh, int const nC, float const dc){
    int iA = (blockIdx.x + blockIdx.y * gridDim.x) * CHUNK;    
    int tx = threadIdx.x;
    __shared__ float sA[NC][CHUNK];
    __shared__ float rho1[NTHREADS][CHUNK];
    
    // cache A
    if (tx < nC){ //use tx as iC
        for (int i=0; i<CHUNK; ++i){
            if (iA+i < nA){
                sA[tx][i] = A[tx + (iA+i)*nC];
            }else{
                sA[tx][i] = SINGLE_INF;
            }   
        }
    }
    for (int i=0; i<CHUNK; ++i) rho1[tx][i] = 0.0f;
    __syncthreads();
    
    // fill in the shared memory A
    float dc2 = dc*dc;
    int iB_min = MAX(iA - nneigh, 0);
    int iB_max = MIN(iA + nneigh + CHUNK - 1, nA-1);
    int iB = iB_min + tx; //MAX(tx, iB_min); // tx is index for B    
    while (iB <= iB_max){
        float dist[CHUNK];
        // calculate distance to B        
        for (int i=0; i<CHUNK; ++i) dist[i] = 0.0f;
        for (int iC=0; iC<nC; ++iC){
            float Btemp = A[iC + iB*nC];
            for (int i=0; i<CHUNK; ++i){
                float temp = Btemp - sA[iC][i];
                dist[i] += temp * temp;
            }            
        }          
        for (int i=0; i<CHUNK; ++i){            
            int dab = ABS(iA+i-iB);
            if (dab<=nneigh){
                if (iA+i < nA && iA+i != iB){
                    rho1[tx][i] += expf(-1*dist[i]/dc2);
                }
            }
        }
        iB += blockDim.x;
    } // while
    
    // final count
    __syncthreads();    
    // if (tx < CHUNK) D[iA+tx] = rho1[tx];
    if (tx < CHUNK){
        float sum = 0.0f;
        for (int tx1=0; tx1<blockDim.x; ++tx1)
            sum += rho1[tx1][tx];
        if (iA+tx<nA) D[iA+tx] = sum;
    }
} // func